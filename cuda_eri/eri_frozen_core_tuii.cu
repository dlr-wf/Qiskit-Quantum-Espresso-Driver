#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <fstream>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/complex.h>

// 64 Threads per block if we calculate one matrix element per BLOCK
constexpr int ThreadsPerBlock = 512;

// blockDim.x,y,z gives the number of threads in a block, in the particular direction
// gridDim.x,y,z gives the number of blocks in a grid, in the particular direction
// blockDim.x * gridDim.x gives the number of threads in a grid (in the x direction, in this case)

// NVIDIA A100 Information (https://docs.nvidia.com/cuda/ampere-tuning-guide/index.html):
// maximum number of thread blocks per SM is 32

__constant__ float PI = 3.14159265358979323846;

__global__ void sum_p_reduction_tuii(const thrust::complex<double>* c_ip, const double* p_vec, thrust::complex<double>* output,
                                     const int c_ip_size, const int p_size, const int n_bands_active, const int n_bands_core, const int n_waves) {
    // Calculates h_tuii where t,u are active space indices and i is a core index
    
    // Shapes:
    // c_ip: (#waves * #states), n_waves is fastest changing index
    // p: (#waves * 3), 3 (coordinates) is fastest changing index

    // TODO: Use more threads and calculate multiple matrix elements per block, e.g. different i indices
    //       Instead of using the block index to describe the i index. Maybe no real speed up here.

    // int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int blockId = blockIdx.x;
    int threadId = threadIdx.x;
    double abs_error = 1e-3;
    
    if (blockId < n_bands_active * n_bands_active * n_bands_core) {
        int t = blockId % n_bands_active;
        int u = (blockId / n_bands_active) % n_bands_active;
        int i = (blockId / (n_bands_active * n_bands_active)) % n_bands_core;

        __shared__ thrust::complex<double> partialSum[ThreadsPerBlock];
        partialSum[threadId] = thrust::complex<double>(0.0f, 0.0f);

        #pragma unroll
        for (int p_index = threadId; p_index < n_waves; p_index += blockDim.x) {
            // c_ip of i-th band of the first wave has index i*n_waves, ... of 2nd wave has index i*n_waves+1, ... of last wave has index (i+1)*n_waves
            double p_x = p_vec[p_index*3+0];
            double p_y = p_vec[p_index*3+1];
            double p_z = p_vec[p_index*3+2];

            thrust::complex<double> c_ip_conj = thrust::conj(c_ip[t*n_waves+p_index]);

            #pragma unroll
            for (int q_index = 0; q_index < n_waves; ++q_index) {
                double q_x = p_vec[q_index*3+0];
                double q_y = p_vec[q_index*3+1];
                double q_z = p_vec[q_index*3+2];

                thrust::complex<double> c_jq_conj = thrust::conj(c_ip[u*n_waves+q_index]);

                #pragma unroll
                for (int s_index = 0; s_index < n_waves; ++s_index) {
                    double s_x = p_vec[s_index*3+0];
                    double s_y = p_vec[s_index*3+1];
                    double s_z = p_vec[s_index*3+2];

                    #pragma unroll
                    for (int r_index = 0; r_index < n_waves; ++r_index) {
                        if (p_index == s_index) {
                            continue;
                        }

                        double r_x = p_vec[r_index*3+0];
                        double r_y = p_vec[r_index*3+1];
                        double r_z = p_vec[r_index*3+2];

                        thrust::complex<double> four_pi_over_p_minus_s_squared = thrust::complex<double>(4.0f * PI /
                                                                                                        ((p_x - s_x)*(p_x - s_x) +
                                                                                                         (p_y - s_y)*(p_y - s_y) +
                                                                                                         (p_z - s_z)*(p_z - s_z)), 0.0f);

                        bool p_minus_r_equals_s_minus_q = (std::abs(p_x - r_x - (s_x - q_x)) <= abs_error &&
                                                           std::abs(p_y - r_y - (s_y - q_y)) <= abs_error &&
                                                           std::abs(p_z - r_z - (s_z - q_z)) <= abs_error);

                        if (p_minus_r_equals_s_minus_q) {
                            partialSum[threadId] += c_ip_conj
                                * c_jq_conj
                                * c_ip[i*n_waves+r_index]
                                * c_ip[i*n_waves+s_index]
                                * four_pi_over_p_minus_s_squared;
                        }
                    }
                }
            }
        }

        // Synchronize the threads within the block to ensure all partial sums are computed
        __syncthreads();

        // Perform reduction to calculate the final sum
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            if (threadId < stride) {
                partialSum[threadId] += partialSum[threadId + stride];
            }
            __syncthreads();
        }

        // Store the final sum in global memory
        if (threadId == 0) {
            output[blockId] = partialSum[0];
            // if (partialSum[0].real() != 0.0 && partialSum[0].imag() != 0.0) {
            //     printf("real=%4.2f, imag=%4.2f \n", partialSum[0].real(), partialSum[0].imag());
            // }
        }
    }
}

std::tuple<int, int, std::vector<std::complex<double>>> load_coeff(std::string filename) {
    std::ifstream inputFile(filename);
    std::vector<double> coefficients_real;
    std::vector<double> coefficients_imag;

    if (!inputFile)
    {
        std::cerr << "Error opening file " << filename << "!" << std::endl;
        throw std::runtime_error("Exception: Error opening file!");
    }

    int n_bands;
    int n_waves;
    inputFile >> n_bands >> n_waves; // n_waves is fastest changing index

    double real;
    double imag;
    while (inputFile >> real >> imag)
    {
        coefficients_real.push_back(real);
        coefficients_imag.push_back(imag);
    }
    inputFile.close();

    int numCoefficients = coefficients_real.size();
    
    assert((numCoefficients==n_waves*n_bands) && "Number of coefficients does not match number of bands times number of waves written in first line");
    
    std::vector<std::complex<double>> c_ip;
    for (size_t i = 0; i < numCoefficients; ++i) {
        std::complex<double> complexNumber(coefficients_real[i], coefficients_imag[i]);
        c_ip.push_back(complexNumber);
    }

    return {n_bands, n_waves, c_ip};
}

std::vector<double> load_p(std::string filename) {
    std::ifstream inputFile(filename);
    std::vector<double> p_x;
    std::vector<double> p_y;
    std::vector<double> p_z;

    if (!inputFile)
    {
        std::cerr << "Error opening file " << filename << "!" << std::endl;
        throw std::runtime_error("Exception: Error opening file!");
    }

    int n_waves;
    inputFile >> n_waves;

    double x, y, z;
    while (inputFile >> x >> y >> z)
    {
        p_x.push_back(x);
        p_y.push_back(y);
        p_z.push_back(z);
    }
    inputFile.close();

    int numWaves = p_x.size();

    assert((numWaves==n_waves) && "Number of waves written in first line does not match actual number of waves");

    std::vector<double> p;
    for (size_t i = 0; i < numWaves; ++i) {
        p.push_back(p_x[i]);
        p.push_back(p_y[i]);
        p.push_back(p_z[i]);
    }

    return p;
}

std::vector<float> load_occ(std::string filename) {
    std::ifstream inputFile(filename);
    std::vector<float> occs;

    if (!inputFile)
    {
        std::cerr << "Error opening file " << filename << "!" << std::endl;
        throw std::runtime_error("Exception: Error opening file!");
    }

    int n_bands;
    inputFile >> n_bands;

    float occ;
    while (inputFile >> occ)
    {
        occs.push_back(occ);
    }
    inputFile.close();

    int numBands = occs.size();

    assert((numBands==n_bands) && "Number of bands written in first line does not match actual number of bands");

    return occs;
}

bool write_output(thrust::host_vector<thrust::complex<float>> output, std::string filename,
                  int n_bands_active, int n_bands_core) {
    // Open the file for writing
    std::ofstream outFile(filename);

    if (!outFile.is_open()) {
        std::cerr << "Unable to open the file." << std::endl;
        return false;
    }

    // Write the number of complex elements to the file
    size_t numComplexElements = output.size();
    outFile << numComplexElements << " " << n_bands_active << " " << n_bands_core << "\n";

    // Write each complex number to the file
    outFile << std::setprecision(std::numeric_limits<float>::max_digits10);

    
    for (int idx=0; idx<output.size(); ++idx) {
        int t = idx % n_bands_active;
        int u = (idx / n_bands_active) % n_bands_active;
        int i = (idx / (n_bands_active * n_bands_active)) % n_bands_core;
        outFile << t << " " << u << " " << i << " " << i << " " << output[idx].real() << " " << output[idx].imag() << "\n";
    }

    // Close the file
    outFile.close();
    std::cout << "Data has been written to the file successfully." << std::endl;

    return true;
}

int main(int argc, char* argv[]) {
    std::string base_folder = "../eri/";
    
    if (argc != 5) {
        std::cerr << "Number of arguments needs to be four, "
                  << "active space start band (int), active space end band (int), "
                  << "frozen core start band (int), frozen core end band (int), but is " << argc-1 << "!" << std::endl;
        return 1;
    }
    std::cout << "Calculating frozen core effective potential component g_tuii" << std::endl;
    // Load coefficients and momentum vectors from file
    std::string filename_coeff {base_folder+"c_ip.txt"};
    std::string filename_p {base_folder+"p.txt"};
    std::string filename_occ {base_folder+"occ_binary.txt"};
    std::tuple<int, int, std::vector<std::complex<double>>> coeff_tuple = load_coeff(filename_coeff);
    int n_bands_all = std::get<0>(coeff_tuple);
    int n_waves_all = std::get<1>(coeff_tuple);
    std::vector<std::complex<double>> c_ip_all = std::get<2>(coeff_tuple);
    std::vector<double> p_all = load_p(filename_p);
    std::vector<float> occ = load_occ(filename_occ);

    std::cout << "occ size: " << occ.size() << std::endl;

    int start_band_active = std::stoi(argv[1]); // included
    int end_band_active = std::stoi(argv[2]);   // excluded
    int start_band_core = std::stoi(argv[3]); // included
    int end_band_core = std::stoi(argv[4]);   // excluded
    int n_bands_active = end_band_active-start_band_active;
    int n_bands_core = end_band_core-start_band_core;
    if (start_band_active < 0 || start_band_active >= n_bands_all) {
        std::cerr << "Active space start band (" << start_band_active << ") is invalid! Needs to be 0 or larger and must not be equal or larger than "
                  << n_bands_all << std::endl;
        return 1;
    }
    if (end_band_active < 0 || end_band_active > n_bands_all || end_band_active <= start_band_active) {
        std::cerr << "Active space end band (" << end_band_active << ") is invalid! Needs to be 0 or larger and must not be equal or larger than "
                  << n_bands_all << " and cannot be equal or smaller than active space start band (" << start_band_active << ")!" << std::endl;
        return 1;
    }
    if (start_band_core < 0 || start_band_core >= n_bands_all) {
        std::cerr << "Frozen core start band (" << start_band_core << ") is invalid! Needs to be 0 or larger and must not be equal or larger than "
                  << n_bands_all << std::endl;
        return 1;
    }
    if (end_band_core < 0 || end_band_core > n_bands_all || end_band_core <= start_band_core) {
        std::cerr << "Frozen core end band (" << end_band_core << ") is invalid! Needs to be 0 or larger and must not be equal or larger than "
                  << n_bands_all << " and cannot be equal or smaller than frozen core start band (" << start_band_core << ")!" << std::endl;
        return 1;
    }
    if (end_band_core > start_band_active) {
        std::cerr << "Frozen core end band (" << end_band_core << ") is invalid! Needs to be smaller or equal to active space start band ("
                  << start_band_active << ")!" << std::endl;
        return 1;
    }

    std::cout << "Occupations of bands (selected active [" << start_band_active << ", " << end_band_active << "] core ["
              << start_band_core << ", " << end_band_core << "] in |...|): ";
    for (int i = 0; i < occ.size(); ++i) {
        if (i == start_band_core || i == end_band_core || i == start_band_active || i == end_band_active) {
            std::cout << "| ";
        }
        std::cout << occ[i] << " ";
    }
    if (end_band_core == occ.size() || end_band_active == occ.size()) {
        std::cout << "|";
    }
    std::cout << std::endl;

    int n_waves = n_waves_all;

    std::vector<std::complex<float>> c_ip_active = std::vector<std::complex<float>>(c_ip_all.begin()+start_band_active*n_waves_all,
                                                                                    c_ip_all.begin()+end_band_active*n_waves_all);
    std::vector<std::complex<float>> c_ip_bands = std::vector<std::complex<float>>(c_ip_all.begin()+start_band_core*n_waves_all,
                                                                                   c_ip_all.begin()+end_band_core*n_waves_all);
    c_ip_bands.insert(c_ip_bands.end(), c_ip_active.begin(), c_ip_active.end()); // Concatenate core and active space coefficients, c_ip_bands = [(c_ip_core), (c_ip_active)]
    
    std::vector<std::complex<double>> c_ip;
    c_ip.reserve((n_bands_core+n_bands_active)*n_waves);
    for (int i=0; i<c_ip_bands.size(); ++i) {
        if ((i%n_waves_all)<n_waves) {
            c_ip.push_back(c_ip_bands[i]);
        }
    }

    std::vector<double> p = std::vector<double>(p_all.begin(),
                                                p_all.begin()+n_waves*3);

    std::cout << "n_bands_core: " << n_bands_core << ", n_bands_active: " << n_bands_active << std::endl;
    std::cout << "n_waves: " << n_waves << ", n_waves_all: " << n_waves_all << std::endl;

    size_t size_c_ip = c_ip.size();
    thrust::device_vector<thrust::complex<double>> dev_c_ip = c_ip;
    std::cout << "size_c_ip: " << size_c_ip << std::endl;

    size_t size_p = p.size();
    thrust::device_vector<double> dev_p = p;
    std::cout << "size_p: " << size_p << std::endl;

    int output_size = n_bands_active*n_bands_active*n_bands_core;
    thrust::device_vector<thrust::complex<double>> devOutput(output_size, thrust::complex<double>(0.0f, 0.0f));

    // Define the block and grid sizes
    int blocks = output_size;
    
    // Launch the kernel
    std::cout << "CUDA kernel launch with " << blocks << " blocks of " << ThreadsPerBlock  << " threads!" << std::endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    sum_p_reduction_tuii<<<blocks, ThreadsPerBlock>>>(thrust::raw_pointer_cast(dev_c_ip.data()), // Takes ~4min for one band
                                                      thrust::raw_pointer_cast(dev_p.data()),
                                                      thrust::raw_pointer_cast(devOutput.data()),
                                                      size_c_ip, size_p, n_bands_active, n_bands_core, n_waves);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n",
            hipGetErrorString(cudaerr));
        return 1;
    }

    cudaerr = hipPeekAtLastError();
    if (cudaerr != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n",
            hipGetErrorString(cudaerr));
        return 1;
    }
    hipEventRecord(stop);
    
    // Copy the result back to the host
    thrust::host_vector<thrust::complex<double>> hostOutput = devOutput;

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Elapsed CUDA Time [ms]/[s]/[min]: " << milliseconds << "/" << milliseconds/1000 << "/" << milliseconds/1000/60 << std::endl;

    // Print the result
    std::cout << "output size: " << hostOutput.size() << std::endl;

    std::string filename_output {base_folder+"eri_frozen_tuii_active_"+std::to_string(start_band_active)+"_"+std::to_string(end_band_active)
                                 +"_core_"+std::to_string(start_band_core)+"_"+std::to_string(end_band_core)+".txt"};
    bool writeOutput = write_output(hostOutput, filename_output, n_bands_active, n_bands_core);
    if (writeOutput) {
        std::cout << "Electron repulsion integrals successfully written to " << filename_output << std::endl;
    }
    else {
        std::cout << "Could not write output to file!" << std::endl;
    }

    std::cout << "Finished!" << std::endl;

    return 0;
}
